#include "hip/hip_runtime.h"
// Histogram Equalization

#include    <wb.h>
#include <hip/hip_runtime.h>

#define HISTOGRAM_LENGTH 256

//#define RUN_ON_HOST

#define WEIGHT_R   0.21f
#define WEIGHT_G   0.71f
#define WEIGHT_B   0.07f

typedef unsigned char UCHAR, *PUCHAR;
typedef unsigned int UINT, *PUINT;

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", err);                        \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return err;                                                       \
        }                                                                     \
    } while(0)

//@@ insert code here

#if defined(RUN_ON_HOST)

hipError_t
castToUChar(const float *floatImage, PUCHAR ucharImage, int height, int width, int channels)
{
    for(int i=0;i<height*width*channels;i++)
    {
        ucharImage[i] = (unsigned char)(255.0f * floatImage[i]);
    }

    return hipSuccess;
}

hipError_t
convertRGBToGray(const PUCHAR rgbImage, PUCHAR grayImage, int height, int width, int channels)
{
    assert(channels == 3);

    for(int y=0;y<height;y++)
    {
        for(int x=0;x<width;x++)
        {
            int index = y*width + x;

            float r = rgbImage[channels*index + 0];
            float g = rgbImage[channels*index + 1];
            float b = rgbImage[channels*index + 2];

            grayImage[index] = (UCHAR)(r*WEIGHT_R + g*WEIGHT_G + b*WEIGHT_B);
        }
    }

    return hipSuccess;
}

UCHAR
clamp(const UCHAR x, const UCHAR start, const UCHAR end)
{
    return min(max(x,start), end);
}

UCHAR
correctColor(UCHAR val, float *cdf)
{
    return clamp((UCHAR)(255*(cdf[val]-cdf[0])/(1-cdf[0])), 0, 255);
}

hipError_t
correctColors(PUCHAR ucharImage, float *cdf, int height, int width, int channels)
{
    for(int i=0;i<height*width*channels;i++)
    {
        ucharImage[i] = correctColor(ucharImage[i], cdf);
    }

    return hipSuccess;
}

hipError_t
castToFloat(const PUCHAR input, float *output, int height, int width, int channels)
{
    for(int i=0;i<height*width*channels;i++)
    {
        output[i] = (float)(input[i]/255.0f);
    }

    return hipSuccess;
}

#else

__global__
void gpu_castToUChar(const float *floatImage, PUCHAR ucharImage, int height, int width, int channels)
{
	const int ty = threadIdx.y;
	const int tx = threadIdx.x;
    const int idx = (blockIdx.y + ty)*width + blockIdx.x*blockDim.x + tx;

    if (idx < height*width)
    {
        for(int i=0;i<channels;i++)
        {
            int index = idx*3 + i;
            ucharImage[index] = (UCHAR)(255.0f * floatImage[index]);
        }
    }
}

hipError_t
castToUChar(const float *floatImage, PUCHAR ucharImage, int height, int width, int channels)
{
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1+(width-1)/dimBlock.x, 1+(height-1)/dimBlock.y, 1);

    float *gpu_floatImage=NULL;
    PUCHAR gpu_ucharImage=NULL;

    wbCheck(hipHostGetDevicePointer((void **)&gpu_floatImage, (void *)floatImage, 0));
    wbCheck(hipHostGetDevicePointer((void **)&gpu_ucharImage, (void *)ucharImage, 0));

    gpu_castToUChar<<<dimGrid, dimBlock>>>(gpu_floatImage, gpu_ucharImage, height, width, channels);

    wbCheck(hipDeviceSynchronize());

    return hipSuccess;
}

__global__
void gpu_convertRGBToGray(const PUCHAR rgbImage, PUCHAR grayImage, int height, int width, int channels)
{
	const int ty = threadIdx.y;
	const int tx = threadIdx.x;
    const int idx = (blockIdx.y + ty)*width + blockIdx.x*blockDim.x + tx;

    if (idx < height*width)
    {
        int index = idx*3;

        float r = rgbImage[channels*index + 0];
        float g = rgbImage[channels*index + 1];
        float b = rgbImage[channels*index + 2];

        grayImage[idx] = (UCHAR)(r*WEIGHT_R + g*WEIGHT_G + b*WEIGHT_B);
    }
}

hipError_t
convertRGBToGray(const PUCHAR rgbImage, PUCHAR grayImage, int height, int width, int channels)
{
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1+(width-1)/dimBlock.x, 1+(height-1)/dimBlock.y, 1);

    PUCHAR gpu_rgbImage=NULL;
    PUCHAR gpu_grayImage=NULL;

    assert(channels == 3);

    wbCheck(hipHostGetDevicePointer((void **)&gpu_rgbImage, (void *)rgbImage, 0));
    wbCheck(hipHostGetDevicePointer((void **)&gpu_grayImage, (void *)grayImage, 0));

    gpu_convertRGBToGray<<<dimGrid, dimBlock>>>(gpu_rgbImage, gpu_grayImage, height, width, channels);

    wbCheck(hipDeviceSynchronize());

    return hipSuccess;
}

__device__
UCHAR
clamp(const UCHAR x, const UCHAR start, const UCHAR end)
{
    return min(max(x,start), end);
}

__device__
UCHAR
correctColor(UCHAR val, float *cdf)
{
    return clamp((UCHAR)(255*(cdf[val]-cdf[0])/(1-cdf[0])), 0, 255);
}

__global__
void gpu_correctColors(PUCHAR ucharImage, float *cdf, int height, int width, int channels)
{
	const int ty = threadIdx.y;
	const int tx = threadIdx.x;
    const int idx = (blockIdx.y + ty)*width + blockIdx.x*blockDim.x + tx;

    if (idx < height*width)
    {
        for(int i=0;i<channels;i++)
        {
            int index = idx*3 + i;
            ucharImage[index] = correctColor(ucharImage[index], cdf);
        }
    }
}

hipError_t
correctColors(PUCHAR ucharImage, float *cdf, int height, int width, int channels)
{
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1+(width-1)/dimBlock.x, 1+(height-1)/dimBlock.y, 1);

    PUCHAR gpu_ucharImage=NULL;
    float *gpu_cdf=NULL;

    wbCheck(hipHostGetDevicePointer((void **)&gpu_ucharImage, (void *)ucharImage, 0));
    wbCheck(hipHostGetDevicePointer((void **)&gpu_cdf, (void *)cdf, 0));

    gpu_correctColors<<<dimGrid, dimBlock>>>(gpu_ucharImage, gpu_cdf, height, width, channels);

    wbCheck(hipDeviceSynchronize());

    return hipSuccess;
}

__global__
void gpu_castToFloat(const PUCHAR ucharImage, float *floatImage, int height, int width, int channels)
{
	const int ty = threadIdx.y;
	const int tx = threadIdx.x;
    const int idx = (blockIdx.y + ty)*width + blockIdx.x*blockDim.x + tx;

    if (idx < height*width)
    {
        for(int i=0;i<channels;i++)
        {
            int index = idx*3 + i;
            floatImage[index] = (float)(ucharImage[index]/255.0f);
        }
    }
}

hipError_t
castToFloat(const PUCHAR ucharImage, float *floatImage, int height, int width, int channels)
{
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1+(width-1)/dimBlock.x, 1+(height-1)/dimBlock.y, 1);

    PUCHAR gpu_ucharImage=NULL;
    float *gpu_floatImage=NULL;

    wbCheck(hipHostGetDevicePointer((void **)&gpu_ucharImage, (void *)ucharImage, 0));
    wbCheck(hipHostGetDevicePointer((void **)&gpu_floatImage, (void *)floatImage, 0));

    gpu_castToFloat<<<dimGrid, dimBlock>>>(gpu_ucharImage, gpu_floatImage, height, width, channels);

    wbCheck(hipDeviceSynchronize());

    return hipSuccess;
}

#endif

#if defined(RUN_ON_HOST)
hipError_t
computeHistogram(const PUCHAR grayImage, PUINT histogram, int height, int width)
{
    for(int i=0;i<HISTOGRAM_LENGTH;i++)
    {
        histogram[i] = 0;
    }

    for(int i=0;i<height*width;i++)
    {
        histogram[grayImage[i]]++;
    }

    return hipSuccess;
}

#else

__global__
void gpu_computeHistogram(const PUCHAR grayImage, PUINT histogram, int height, int width)
{
	const int tx = threadIdx.x;
    const int idx = blockIdx.x*blockDim.x + tx;
    const int stride = blockDim.x * gridDim.x;

    __shared__ UINT privateHistogram[HISTOGRAM_LENGTH];

    for(int i = tx; i < HISTOGRAM_LENGTH; i += blockDim.x)
    {
        privateHistogram[i] = 0;
    }

    __syncthreads();

    while (idx < height * width)
    {
        atomicAdd(&privateHistogram[grayImage[idx]], 1);
        idx += stride;
    }

    __syncthreads();

    for(int i = tx; i < HISTOGRAM_LENGTH; i += blockDim.x)
    {
        atomicAdd(&histogram[i], privateHistogram[i]);
    }
}

hipError_t
computeHistogram(const PUCHAR grayImage, PUINT histogram, int height, int width)
{
    dim3 dimBlock(1, 1, 1);
    dim3 dimGrid(1+(width+height-1)/dimBlock.x, 1, 1);

    PUCHAR gpu_grayImage=NULL;
    PUINT  gpu_histogram=NULL;

    wbCheck(hipHostGetDevicePointer((void **)&gpu_grayImage, (void *)grayImage, 0));
    wbCheck(hipHostGetDevicePointer((void **)&gpu_histogram, (void *)histogram, 0));

    gpu_computeHistogram<<<dimGrid, dimBlock>>>(gpu_grayImage, gpu_histogram, height, width);

    wbCheck(hipDeviceSynchronize());

    return hipSuccess;
}

#endif

hipError_t
computeCDF(const PUINT histogram, float *cdf, int height, int width)
{
    float prev = 0.0;

    for(int i=0;i<HISTOGRAM_LENGTH;i++)
    {
        cdf[i] = prev + histogram[i]/(float)(height*width);
        prev = cdf[i];
    }

    return hipSuccess;
}

hipError_t
equalizeHistogram(const float *inputData, float * outputData, int height, int width, int channels)
{
    float  *floatImage = NULL;
    PUCHAR ucharImage  = NULL;
    PUCHAR grayImage   = NULL;
    PUINT  histogram   = NULL;
    float *cdf         = NULL;

    wbCheck(hipHostAlloc((void **)&floatImage, sizeof(float) * height * width * channels, hipHostMallocMapped));
    wbCheck(hipHostAlloc((void **)&ucharImage, sizeof(UCHAR) * height * width * channels, hipHostMallocMapped));
    wbCheck(hipHostAlloc((void **)&grayImage, sizeof(UCHAR) * height * width, hipHostMallocMapped));
    wbCheck(hipHostAlloc((void **)&histogram, sizeof(UINT) * HISTOGRAM_LENGTH, hipHostMallocMapped));
    wbCheck(hipHostAlloc((void **)&cdf, sizeof(float) * HISTOGRAM_LENGTH, hipHostMallocMapped));

    memcpy(floatImage, inputData, sizeof(float) * height * width * channels);

    wbCheck(castToUChar(floatImage, ucharImage, height, width, channels));
    wbCheck(convertRGBToGray(ucharImage, grayImage, height, width, channels));
    wbCheck(computeHistogram(grayImage, histogram, height, width));
    wbCheck(computeCDF(histogram, cdf, height, width));
    wbCheck(correctColors(ucharImage, cdf, height, width, channels));
    wbCheck(castToFloat(ucharImage, floatImage, height, width, channels));

    memcpy(outputData, floatImage, sizeof(float) * height * width * channels);

    hipHostFree(cdf);
    hipHostFree(histogram);
    hipHostFree(grayImage);
    hipHostFree(ucharImage);

    return hipSuccess;
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int imageWidth;
    int imageHeight;
    int imageChannels;
    wbImage_t inputImage;
    wbImage_t outputImage;
    const char * inputImageFile;

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);

    wbTime_start(Generic, "Importing data and creating memory on host");
    inputImage = wbImport(inputImageFile);
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hipDeviceProp_t prop;
    int whichDevice;

    wbCheck(hipGetDevice(&whichDevice));
    wbCheck(hipGetDeviceProperties(&prop, whichDevice));

    wbLog(TRACE, "prop.canMapHostMemory=", prop.canMapHostMemory);

    wbCheck(hipSetDeviceFlags(hipDeviceMapHost));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    //@@ insert code here
    wbLog(TRACE, "The image height is ", imageHeight);
    wbLog(TRACE, "The image width is ",  imageWidth);
    wbLog(TRACE, "The number of channels is ",  imageChannels);
    wbLog(TRACE, "The number of pixels is ",  imageHeight*imageWidth*imageChannels);

    wbCheck(equalizeHistogram(wbImage_getData(inputImage),
                              wbImage_getData(outputImage),
                              imageHeight,
                              imageWidth,
                              imageChannels));

    wbSolution(args, outputImage);

    //@@ insert code here
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}

