#include "hip/hip_runtime.h"
#include	<wb.h>

#define CHUNK_SIZE  512
#define BLOCK_SIZE   64

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", err);                        \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    //@@ Insert code to implement vector addition here
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	if (idx < len)
	{
		out[idx] = in1[idx] + in2[idx];
	}
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;

    float * pinnedHostInput1;
    float * pinnedHostInput2;
    float * pinnedHostOutput;

    float * deviceInput0_1;
    float * deviceInput0_2;
    float * deviceOutput0;

    float * deviceInput1_1;
    float * deviceInput1_2;
    float * deviceOutput1;

    float * deviceInput2_1;
    float * deviceInput2_2;
    float * deviceOutput2;

    float * deviceInput3_1;
    float * deviceInput3_2;
    float * deviceOutput3;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

	wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here

    wbCheck(hipHostAlloc((void **)&pinnedHostInput1, inputLength * sizeof(float), hipHostMallocDefault));
    wbCheck(hipHostAlloc((void **)&pinnedHostInput2, inputLength * sizeof(float), hipHostMallocDefault));
    wbCheck(hipHostAlloc((void **)&pinnedHostOutput, inputLength * sizeof(float), hipHostMallocDefault));

	wbCheck(hipMalloc((void **)&deviceInput0_1, CHUNK_SIZE * sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceInput0_2, CHUNK_SIZE * sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceOutput0, CHUNK_SIZE * sizeof(float)));

	wbCheck(hipMalloc((void **)&deviceInput1_1, CHUNK_SIZE * sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceInput1_2, CHUNK_SIZE * sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceOutput1, CHUNK_SIZE * sizeof(float)));

	wbCheck(hipMalloc((void **)&deviceInput2_1, CHUNK_SIZE * sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceInput2_2, CHUNK_SIZE * sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceOutput2, CHUNK_SIZE * sizeof(float)));

	wbCheck(hipMalloc((void **)&deviceInput3_1, CHUNK_SIZE * sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceInput3_2, CHUNK_SIZE * sizeof(float)));
	wbCheck(hipMalloc((void **)&deviceOutput3, CHUNK_SIZE * sizeof(float)));

    wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(Generic, "Creating streams.");

    hipStream_t stream0;
    hipStream_t stream1;
    hipStream_t stream2;
    hipStream_t stream3;

    wbCheck(hipStreamCreate(&stream0));
    wbCheck(hipStreamCreate(&stream1));
    wbCheck(hipStreamCreate(&stream2));
    wbCheck(hipStreamCreate(&stream3));

	wbTime_stop(Generic, "Creating streams.");

    memcpy(pinnedHostInput1, hostInput1, inputLength*sizeof(float));
    memcpy(pinnedHostInput2, hostInput2, inputLength*sizeof(float));

	wbTime_start(Generic, "Queuing items to streams.");
	wbTime_start(GPU, "Running steams.");
    for(int pos=0;pos<inputLength; pos += CHUNK_SIZE)
    {
        int left = min(inputLength-pos, CHUNK_SIZE);
        dim3 DimGrid(1 + (left-1)/BLOCK_SIZE, 1, 1);
        dim3 DimBlock(BLOCK_SIZE, 1, 1);

        wbCheck(hipMemcpyAsync(deviceInput0_1, pinnedHostInput1+pos, left*sizeof(float), hipMemcpyHostToDevice, stream0));
        wbCheck(hipMemcpyAsync(deviceInput0_2, pinnedHostInput2+pos, left*sizeof(float), hipMemcpyHostToDevice, stream0));

        vecAdd<<<DimGrid, DimBlock, 0, stream0>>>(deviceInput0_1, deviceInput0_2, deviceOutput0, left);

        wbCheck(hipMemcpyAsync(pinnedHostOutput+pos, deviceOutput0, left*sizeof(float), hipMemcpyDeviceToHost, stream0));
    }
	wbTime_stop(Generic, "Queuing items to streams.");

	wbTime_start(Generic, "Synchronizing for streams.");
    wbCheck(hipStreamSynchronize(stream0));
    wbCheck(hipStreamSynchronize(stream1));
    wbCheck(hipStreamSynchronize(stream2));
    wbCheck(hipStreamSynchronize(stream3));
	wbTime_stop(Generic, "Synchronizing for streams.");

	wbTime_stop(GPU, "Running steams.");

    memcpy(hostOutput, pinnedHostOutput, inputLength*sizeof(float));

    wbSolution(args, hostOutput, inputLength);

    wbCheck(hipHostFree(pinnedHostInput1));
    wbCheck(hipHostFree(pinnedHostInput2));
    wbCheck(hipHostFree(pinnedHostOutput));

	wbCheck(hipFree(deviceInput0_1));
	wbCheck(hipFree(deviceInput0_2));
	wbCheck(hipFree(deviceOutput0));

	wbCheck(hipFree(deviceInput1_1));
	wbCheck(hipFree(deviceInput1_2));
	wbCheck(hipFree(deviceOutput1));

	wbCheck(hipFree(deviceInput2_1));
	wbCheck(hipFree(deviceInput2_2));
	wbCheck(hipFree(deviceOutput2));

	wbCheck(hipFree(deviceInput3_1));
	wbCheck(hipFree(deviceInput3_2));
	wbCheck(hipFree(deviceOutput3));

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}

